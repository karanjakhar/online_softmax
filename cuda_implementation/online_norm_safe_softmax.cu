#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>


#define CUDA_CHECK(ans)                        \
    {                                          \
        cudaAssert((ans), __FILE__, __LINE__); \
    }
inline void cudaAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA error %s: %s at %s: %d\n",
                hipGetErrorName(code), hipGetErrorString(code),
                file, line);
        exit(code);
    }
}

__global__
void safeSoftmaxOnlineNorm(float * inputMatix,int rows, int cols, float * result){
    
    __shared__ float smem[1024];

    for(int row = blockIdx.x; row < rows; row += gridDim.x){
    int tid = threadIdx.x;

    if(row >= rows) return;

    float * input_row = inputMatix + row * cols;
    float * output_row = result + row * cols; 

    float local_max = -INFINITY;
    float curr_max = 0.0f;
    float local_norm = 0.0f;

    for(int j = tid; j < cols; j += blockDim.x){
        curr_max = fmax(input_row[j], local_max);
        local_norm = local_norm * expf(local_max - curr_max) + expf(input_row[j] - curr_max);
        local_max = curr_max;
        }



    // sync all threads and writing their local max to shared memory
    __syncthreads();

    smem[tid] = local_max;
    // sync again, so all threads complete writing 
    __syncthreads();
    

    // Now we start our reduction step

    for(int stride = blockDim.x / 2; stride > 0; stride /= 2){
        if (tid < stride){
        smem[tid] = fmax(smem[tid], smem[tid + stride]);
        }
        __syncthreads();
    }

    float row_max = smem[0];
    
    __syncthreads();


    // Now we will calculate row norm using reduction

    smem[tid] = local_norm * expf(local_max - row_max);

    __syncthreads();

    for(int stride = blockDim.x / 2; stride > 0; stride /= 2){
        if (tid < stride){

        smem[tid] += smem[tid + stride];
        }
        __syncthreads(); 
    }

    float row_norm = smem[0];

    __syncthreads();

    for(int i = tid; i < cols; i += blockDim.x){
        output_row[i] = expf(input_row[i] - row_max)/ row_norm;
    }
    }

}

void print_array(float * arr, int num_elements){
    

    for (int i = 0; i < num_elements; i++){
        printf("%f\t",arr[i]);
    }
    printf("\n");
}

void check_softmax_prob_sum(float * arr, int rows, int cols){
    float sum = 0.f;
    bool checkFailed = false;
    for(int i = 0; i < rows; i++){
        sum = 0.f;
        for(int j = 0; j < cols; j++){
            sum += arr[i*cols+j];
            // printf("sum: %f\n", sum);

        }
        if(sum > 1.2 || sum < 0.9){

            checkFailed = true;
            printf("Softmax check failed : sum of %d is %f \n", i+1, sum);
            return;
        }
        
        // printf("Sum for %d row:%f\n",i, sum );
    }
    if(!checkFailed) printf("Softmax check passed!!\n");
}


/*

The function uses the Box-Muller transform to convert two 
uniformly distributed random numbers (u1 and u2) into a 
normally distributed number (num) with mean 0 and std 1.


*/
float random_normal_clamped(float min, float max) {
    float u1 = (float)rand() / RAND_MAX;
    float u2 = (float)rand() / RAND_MAX;
    float num = sqrtf(-2.0f * logf(u1)) * cosf(2.0f * M_PI * u2);
    if (num < min)
        return min;
    if (num > max)
        return max;
    return num;
}


int main(){
    
    


    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
   
    int rows = 1024, cols = 32768;
    int matrixSize = rows * cols;
    size_t totalBytes = matrixSize * sizeof(float);
    float * inputMatrix = (float*)malloc(totalBytes);
    float * resultMatrix = (float*)malloc(totalBytes);

    if (inputMatrix == NULL){
        printf("Memory allocation failed\n");
        return;
    }


    printf("Num elements:%d\n", matrixSize);
    
    for(int i = 0; i < matrixSize; i++){
        inputMatrix[i] = random_normal_clamped(-10, 10);
    }
    
    

    float * inputMatrix_d, * resultMatrix_d;

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.0f;

    hipEventRecord(start);
    CUDA_CHECK(hipMalloc(&inputMatrix_d, totalBytes));
    CUDA_CHECK(hipMalloc(&resultMatrix_d, totalBytes));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("GPU allocation time: %f ms\n", ms);

    hipEventRecord(start);
    hipMemcpy(inputMatrix_d, inputMatrix, totalBytes, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Host to device transfer time: %f ms\n", ms);

    hipEventRecord(start);
    int threadsPerBlock = maxThreadsPerBlock;
    int blocksPerGrid = 1024;
    printf("threadsPerBlock: %d\n", threadsPerBlock);
    safeSoftmaxOnlineNorm<<<blocksPerGrid,threadsPerBlock>>>(inputMatrix_d,rows, cols, resultMatrix_d);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time: %f ms\n", ms);
    
    
    hipEventRecord(start);
    hipMemcpy(resultMatrix, resultMatrix_d, totalBytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Device to host transfer time: %f ms\n", ms);





    check_softmax_prob_sum(resultMatrix, rows, cols);

    free(inputMatrix);
    free(resultMatrix);
    hipFree(inputMatrix_d);
    hipFree(resultMatrix_d);

    return 0;
}

